// -----------------------------------------------------------------------------
/**
 * * Name:       main.cu
 * * Purpose:    Driver for K-Means Clustering on GPU
 * * History:    Titouan Le Moan & Max Bedel, 2024
 */
// -----------------------------------------------------------------------------

#include <chrono>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <stdexcept>

#include <hip/hip_runtime.h>

#include "args.hxx"
#include "csv.h"

#include "../utils/kmeans.hxx"

#include "kmeans.cuh"

#define check_out 1

#define REAL float
#define BLOCK_SIZE 32

/*----------------------------------------------------------------------------*/
/* Toplevel driver                                                            */
/*----------------------------------------------------------------------------*/
int main(int argc, char* argv[]) {
    std::cout << "[K-Means Clustering Using GPU]" << std::endl;
    
    // Define parser
    args::ArgumentParser parser("K-Means Clustering Application", "Clusters data using K-Means algorithm.");
    args::HelpFlag help(parser, "help", "Display this help menu", {'h', "help"});
    args::ValueFlag<int> clustersFlag(parser, "clusters", "Number of clusters (k)", {'k', "clusters"}, 120);
    args::ValueFlag<int> maxItersFlag(parser, "max_iters", "Maximum number of iterations", {'m', "max_iters"}, 100);
    args::ValueFlag<std::string> dataFolderFlag(parser, "folder", "The folder where to take the data", {'f', "folder"}, "pad");

    // Parse command-line arguments
    try {
        parser.ParseCLI(argc, argv);
    } catch (const args::Help&) {
        std::cout << parser;
        return EXIT_SUCCESS;
    } catch (const args::ParseError& e) {
        std::cerr << e.what() << std::endl;
        std::cerr << parser;
        return EXIT_FAILURE;
    } catch (const args::ValidationError& e) {
        std::cerr << e.what() << std::endl;
        std::cerr << parser;
        return EXIT_FAILURE;
    }

    // Retrieve argument values
    int k = args::get(clustersFlag);
    int max_iters = args::get(maxItersFlag);
    std::string folder = args::get(dataFolderFlag);

    std::string inputPath = "../data/" + folder + "/data.csv";
    std::string outputPath = "../data/" + folder + "/labels.csv";

    std::cout << " Reading data from " << inputPath << std::endl;
    std::cout << " Writing labels to " << outputPath << std::endl;

    // Load data
    std::vector<Point<REAL>> data;
    try {
        if (folder.compare("pad") == 0) {
            io::CSVReader<3> in(inputPath);
            in.read_header(io::ignore_extra_column, "X", "Y", "Grey");
            REAL x, y;
            int grey;
            while (in.read_row(x, y, grey)) {
                // Since we want to cluster based on the greyscale value, we only keep the points that are grey
                if (grey == 1) {
                    REAL* coords = new REAL[2];
                    coords[0]    = x;
                    coords[1]    = y;

                    Point<REAL> p(coords, 2);
                    data.emplace_back(p);
                }
            }
        } else if (folder.compare("synthetic") == 0) {
            io::CSVReader<3> in(inputPath);
            in.read_header(io::ignore_extra_column, "Feature1", "Feature2", "Feature3");
            REAL x, y, z;
            while (in.read_row(x, y, z)) {
                REAL* coords = new REAL[3];
                coords[0]    = x;
                coords[1]    = y;
                coords[2]    = z;

                Point<REAL> p(coords, 3);
                data.emplace_back(p);
            }
        } else {
            std::cerr << "This folder has no emplementation" << std::endl;
            return EXIT_FAILURE;
        }
    } catch (const std::exception& e) {
        std::cerr << "Error reading data: " << e.what() << std::endl;
        return EXIT_FAILURE;
    }

    size_t M = data.size();
    if (M == 0) {
        std::cerr << "No data loaded. Exiting." << std::endl;
        return EXIT_FAILURE;
    }

    // Setup CUDA environnement 
    hipError_t error;

    hipDeviceProp_t deviceProp;
    int devID = 0;
    error = hipGetDevice(&devID);

    if (error != hipSuccess) {
        printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
    }

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.computeMode == hipComputeModeProhibited) {
        std::cerr << "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice() ." <<std::endl;
        exit(EXIT_SUCCESS);
    }

    if (error != hipSuccess) {
        printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
    } else {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }

    // utilities
    hipEvent_t start;
    hipEvent_t stop;
    float msecTotal;

    // start timer
    hipEventCreate(&start);
    hipEventRecord(start, NULL);

    KMeans<REAL> kmeans(k, max_iters);
    kmeans.fit(data.data(), M, data[0].dimension);
    
    // stop and destroy timer
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);

    /* Performance computation, results and performance printing ------------ */
    std::cout << " == Performances " << std::endl;
    std::cout << "\t Processing time: " << msecTotal << " (ms)" << std::endl;

    if (check_out) {
        int* assignments = kmeans.predict(data.data(), M);
        plotResults(outputPath, assignments, M);
    }

    return EXIT_SUCCESS;
}